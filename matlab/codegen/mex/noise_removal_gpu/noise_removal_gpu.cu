#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * noise_removal_gpu.cu
 *
 * Code generation for function 'noise_removal_gpu'
 *
 */

/* Include files */
#include "MWCudaDimUtility.h"
#include "rt_nonfinite.h"
#include "noise_removal_gpu.h"

/* Function Declarations */
static __global__ void noise_removal_gpu_kernel1(const uint16_T noisyRGB[393216],
  uint16_T inpImg_padded[131072]);
static __global__ void noise_removal_gpu_kernel10(uint16_T inpImg_padded[131841]);
static __global__ void noise_removal_gpu_kernel11(uint16_T inpImg_padded[131072],
  uint16_T b_inpImg_padded[131841]);
static __global__ void noise_removal_gpu_kernel12(uint16_T inpImg_padded[131841],
  uint16_T b_inpImg_padded[131072]);
static __global__ void noise_removal_gpu_kernel13(uint16_T inpImg_padded[131072],
  int32_T initAuxVar, uint16_T rgbFixed[393216]);
static __global__ void noise_removal_gpu_kernel2(uint16_T inpImg_padded[131841]);
static __global__ void noise_removal_gpu_kernel3(uint16_T inpImg_padded[131072],
  uint16_T b_inpImg_padded[131841]);
static __global__ void noise_removal_gpu_kernel4(uint16_T inpImg_padded[131841],
  uint16_T outImg[131072]);
static __global__ void noise_removal_gpu_kernel5(const uint16_T noisyRGB[393216],
  uint16_T inpImg_padded[131072]);
static __global__ void noise_removal_gpu_kernel6(uint16_T inpImg_padded[131841]);
static __global__ void noise_removal_gpu_kernel7(uint16_T inpImg_padded[131072],
  uint16_T b_inpImg_padded[131841]);
static __global__ void noise_removal_gpu_kernel8(uint16_T inpImg_padded[131841],
  uint16_T outImg[131072]);
static __global__ void noise_removal_gpu_kernel9(const uint16_T noisyRGB[393216],
  uint16_T inpImg_padded[131072]);

/* Function Definitions */
static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel1(const
  uint16_T noisyRGB[393216], uint16_T inpImg_padded[131072])
{
  uint32_T threadId;
  int32_T minIdx;
  int32_T maxIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  minIdx = static_cast<int32_T>((threadId % 256U));
  maxIdx = static_cast<int32_T>(((threadId - static_cast<uint32_T>(minIdx)) /
    256U));
  if (maxIdx < 512) {
    inpImg_padded[minIdx + (maxIdx << 8)] = noisyRGB[minIdx + (maxIdx << 8)];
  }
}

static __global__ __launch_bounds__(288, 1) void noise_removal_gpu_kernel10
  (uint16_T inpImg_padded[131841])
{
  uint32_T threadId;
  int32_T minIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  minIdx = static_cast<int32_T>(threadId);
  if (minIdx < 257) {
    inpImg_padded[131584 + minIdx] = static_cast<uint16_T>(0U);
  }
}

static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel11
  (uint16_T inpImg_padded[131072], uint16_T b_inpImg_padded[131841])
{
  uint32_T threadId;
  int32_T maxIdx;
  int32_T minIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  maxIdx = static_cast<int32_T>(threadId);
  if (maxIdx < 512) {
    b_inpImg_padded[256 + 257 * maxIdx] = static_cast<uint16_T>(0U);
    for (minIdx = 0; minIdx < 256; minIdx++) {
      b_inpImg_padded[minIdx + 257 * maxIdx] = inpImg_padded[minIdx + (maxIdx <<
        8)];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void noise_removal_gpu_kernel12
  (uint16_T inpImg_padded[131841], uint16_T b_inpImg_padded[131072])
{
  int32_T maxIdx;
  int32_T iter;
  int32_T i0;
  real_T val;
  uint16_T newIm[4];
  int32_T startIdx;
  int32_T orow;
  int32_T ocol;
  uint16_T t;
  int32_T minIdx;
  int32_T b_iter;
  int32_T c_iter;
  __shared__ uint16_T inpImg_padded_shared[1089];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ocol = mwGetGlobalThreadIndexInYDimension();
  orow = mwGetGlobalThreadIndexInXDimension();
  baseR = orow;
  srow = static_cast<int32_T>(threadIdx.x);
  strideRow = static_cast<int32_T>(blockDim.x);
  scol = static_cast<int32_T>(threadIdx.y);
  strideCol = static_cast<int32_T>(blockDim.y);
  for (y_idx = srow; y_idx <= 32; y_idx += strideRow) {
    baseC = ocol;
    for (x_idx = scol; x_idx <= 32; x_idx += strideCol) {
      if ((static_cast<int32_T>(((static_cast<int32_T>((baseR >= 0))) &&
             (static_cast<int32_T>((baseR < 257)))))) && (static_cast<int32_T>
           (((static_cast<int32_T>((baseC >= 0))) && (static_cast<int32_T>
              ((baseC < 513))))))) {
        inpImg_padded_shared[y_idx + 33 * x_idx] = inpImg_padded[257 * baseC +
          baseR];
      } else {
        inpImg_padded_shared[y_idx + 33 * x_idx] = static_cast<uint16_T>(0U);
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((static_cast<int32_T>((ocol < 512))) && (static_cast<int32_T>((orow < 256))))
  {
    for (maxIdx = 0; maxIdx < 2; maxIdx++) {
      for (i0 = 0; i0 < 2; i0++) {
        newIm[i0 + (maxIdx << 1)] = inpImg_padded_shared[(static_cast<int32_T>
          (threadIdx.x) + i0) + 33 * (static_cast<int32_T>(threadIdx.y) + maxIdx)];
      }
    }

    for (iter = 0; iter < 2; iter++) {
      startIdx = 1 + iter;
      minIdx = startIdx;
      t = newIm[startIdx - 1];
      i0 = 4 - (iter + startIdx);
      for (b_iter = 0; b_iter < i0; b_iter++) {
        c_iter = static_cast<int32_T>(((static_cast<uint32_T>(startIdx) +
          static_cast<uint32_T>(b_iter)) + 1U));
        if (static_cast<int32_T>(newIm[c_iter - 1]) < static_cast<int32_T>(t)) {
          t = newIm[c_iter - 1];
          minIdx = c_iter;
        }
      }

      t = newIm[minIdx - 1];
      newIm[minIdx - 1] = newIm[iter];
      newIm[iter] = t;
      maxIdx = startIdx;
      t = newIm[startIdx - 1];
      i0 = 4 - (iter + startIdx);
      for (b_iter = 0; b_iter < i0; b_iter++) {
        c_iter = static_cast<int32_T>(((static_cast<uint32_T>(startIdx) +
          static_cast<uint32_T>(b_iter)) + 1U));
        if (static_cast<int32_T>(newIm[c_iter - 1]) > static_cast<int32_T>(t)) {
          t = newIm[c_iter - 1];
          maxIdx = c_iter;
        }
      }

      t = newIm[maxIdx - 1];
      newIm[maxIdx - 1] = newIm[3 - iter];
      newIm[3 - iter] = t;
    }

    val = 0.5 * static_cast<real_T>(newIm[1]) + 0.5 * static_cast<real_T>(newIm
      [2]);
    if (val > 0.0) {
      b_inpImg_padded[orow + (ocol << 8)] = static_cast<uint16_T>((val + 0.5));
    } else {
      b_inpImg_padded[orow + (ocol << 8)] = static_cast<uint16_T>(0U);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel13
  (uint16_T inpImg_padded[131072], int32_T initAuxVar, uint16_T rgbFixed[393216])
{
  uint32_T threadId;
  int32_T j;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  j = static_cast<int32_T>(threadId);
  if (j < 131072) {
    rgbFixed[(initAuxVar + j) + 1] = inpImg_padded[j];
  }
}

static __global__ __launch_bounds__(288, 1) void noise_removal_gpu_kernel2
  (uint16_T inpImg_padded[131841])
{
  uint32_T threadId;
  int32_T minIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  minIdx = static_cast<int32_T>(threadId);
  if (minIdx < 257) {
    inpImg_padded[131584 + minIdx] = static_cast<uint16_T>(0U);
  }
}

static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel3
  (uint16_T inpImg_padded[131072], uint16_T b_inpImg_padded[131841])
{
  uint32_T threadId;
  int32_T maxIdx;
  int32_T minIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  maxIdx = static_cast<int32_T>(threadId);
  if (maxIdx < 512) {
    b_inpImg_padded[256 + 257 * maxIdx] = static_cast<uint16_T>(0U);
    for (minIdx = 0; minIdx < 256; minIdx++) {
      b_inpImg_padded[minIdx + 257 * maxIdx] = inpImg_padded[minIdx + (maxIdx <<
        8)];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void noise_removal_gpu_kernel4
  (uint16_T inpImg_padded[131841], uint16_T outImg[131072])
{
  int32_T maxIdx;
  int32_T iter;
  int32_T i0;
  real_T val;
  uint16_T newIm[4];
  int32_T startIdx;
  int32_T orow;
  int32_T ocol;
  uint16_T t;
  int32_T minIdx;
  int32_T b_iter;
  int32_T c_iter;
  __shared__ uint16_T inpImg_padded_shared[1089];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ocol = mwGetGlobalThreadIndexInYDimension();
  orow = mwGetGlobalThreadIndexInXDimension();
  baseR = orow;
  srow = static_cast<int32_T>(threadIdx.x);
  strideRow = static_cast<int32_T>(blockDim.x);
  scol = static_cast<int32_T>(threadIdx.y);
  strideCol = static_cast<int32_T>(blockDim.y);
  for (y_idx = srow; y_idx <= 32; y_idx += strideRow) {
    baseC = ocol;
    for (x_idx = scol; x_idx <= 32; x_idx += strideCol) {
      if ((static_cast<int32_T>(((static_cast<int32_T>((baseR >= 0))) &&
             (static_cast<int32_T>((baseR < 257)))))) && (static_cast<int32_T>
           (((static_cast<int32_T>((baseC >= 0))) && (static_cast<int32_T>
              ((baseC < 513))))))) {
        inpImg_padded_shared[y_idx + 33 * x_idx] = inpImg_padded[257 * baseC +
          baseR];
      } else {
        inpImg_padded_shared[y_idx + 33 * x_idx] = static_cast<uint16_T>(0U);
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((static_cast<int32_T>((ocol < 512))) && (static_cast<int32_T>((orow < 256))))
  {
    for (maxIdx = 0; maxIdx < 2; maxIdx++) {
      for (i0 = 0; i0 < 2; i0++) {
        newIm[i0 + (maxIdx << 1)] = inpImg_padded_shared[(static_cast<int32_T>
          (threadIdx.x) + i0) + 33 * (static_cast<int32_T>(threadIdx.y) + maxIdx)];
      }
    }

    for (iter = 0; iter < 2; iter++) {
      startIdx = 1 + iter;
      minIdx = startIdx;
      t = newIm[startIdx - 1];
      i0 = 4 - (iter + startIdx);
      for (b_iter = 0; b_iter < i0; b_iter++) {
        c_iter = static_cast<int32_T>(((static_cast<uint32_T>(startIdx) +
          static_cast<uint32_T>(b_iter)) + 1U));
        if (static_cast<int32_T>(newIm[c_iter - 1]) < static_cast<int32_T>(t)) {
          t = newIm[c_iter - 1];
          minIdx = c_iter;
        }
      }

      t = newIm[minIdx - 1];
      newIm[minIdx - 1] = newIm[iter];
      newIm[iter] = t;
      maxIdx = startIdx;
      t = newIm[startIdx - 1];
      i0 = 4 - (iter + startIdx);
      for (b_iter = 0; b_iter < i0; b_iter++) {
        c_iter = static_cast<int32_T>(((static_cast<uint32_T>(startIdx) +
          static_cast<uint32_T>(b_iter)) + 1U));
        if (static_cast<int32_T>(newIm[c_iter - 1]) > static_cast<int32_T>(t)) {
          t = newIm[c_iter - 1];
          maxIdx = c_iter;
        }
      }

      t = newIm[maxIdx - 1];
      newIm[maxIdx - 1] = newIm[3 - iter];
      newIm[3 - iter] = t;
    }

    val = 0.5 * static_cast<real_T>(newIm[1]) + 0.5 * static_cast<real_T>(newIm
      [2]);
    if (val > 0.0) {
      outImg[orow + (ocol << 8)] = static_cast<uint16_T>((val + 0.5));
    } else {
      outImg[orow + (ocol << 8)] = static_cast<uint16_T>(0U);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel5(const
  uint16_T noisyRGB[393216], uint16_T inpImg_padded[131072])
{
  uint32_T threadId;
  int32_T minIdx;
  int32_T maxIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  minIdx = static_cast<int32_T>((threadId % 256U));
  maxIdx = static_cast<int32_T>(((threadId - static_cast<uint32_T>(minIdx)) /
    256U));
  if (maxIdx < 512) {
    inpImg_padded[minIdx + (maxIdx << 8)] = noisyRGB[131072 + (minIdx + (maxIdx <<
      8))];
  }
}

static __global__ __launch_bounds__(288, 1) void noise_removal_gpu_kernel6
  (uint16_T inpImg_padded[131841])
{
  uint32_T threadId;
  int32_T minIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  minIdx = static_cast<int32_T>(threadId);
  if (minIdx < 257) {
    inpImg_padded[131584 + minIdx] = static_cast<uint16_T>(0U);
  }
}

static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel7
  (uint16_T inpImg_padded[131072], uint16_T b_inpImg_padded[131841])
{
  uint32_T threadId;
  int32_T maxIdx;
  int32_T minIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  maxIdx = static_cast<int32_T>(threadId);
  if (maxIdx < 512) {
    b_inpImg_padded[256 + 257 * maxIdx] = static_cast<uint16_T>(0U);
    for (minIdx = 0; minIdx < 256; minIdx++) {
      b_inpImg_padded[minIdx + 257 * maxIdx] = inpImg_padded[minIdx + (maxIdx <<
        8)];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void noise_removal_gpu_kernel8
  (uint16_T inpImg_padded[131841], uint16_T outImg[131072])
{
  int32_T maxIdx;
  int32_T iter;
  int32_T i0;
  real_T val;
  uint16_T newIm[4];
  int32_T startIdx;
  int32_T orow;
  int32_T ocol;
  uint16_T t;
  int32_T minIdx;
  int32_T b_iter;
  int32_T c_iter;
  __shared__ uint16_T inpImg_padded_shared[1089];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ocol = mwGetGlobalThreadIndexInYDimension();
  orow = mwGetGlobalThreadIndexInXDimension();
  baseR = orow;
  srow = static_cast<int32_T>(threadIdx.x);
  strideRow = static_cast<int32_T>(blockDim.x);
  scol = static_cast<int32_T>(threadIdx.y);
  strideCol = static_cast<int32_T>(blockDim.y);
  for (y_idx = srow; y_idx <= 32; y_idx += strideRow) {
    baseC = ocol;
    for (x_idx = scol; x_idx <= 32; x_idx += strideCol) {
      if ((static_cast<int32_T>(((static_cast<int32_T>((baseR >= 0))) &&
             (static_cast<int32_T>((baseR < 257)))))) && (static_cast<int32_T>
           (((static_cast<int32_T>((baseC >= 0))) && (static_cast<int32_T>
              ((baseC < 513))))))) {
        inpImg_padded_shared[y_idx + 33 * x_idx] = inpImg_padded[257 * baseC +
          baseR];
      } else {
        inpImg_padded_shared[y_idx + 33 * x_idx] = static_cast<uint16_T>(0U);
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((static_cast<int32_T>((ocol < 512))) && (static_cast<int32_T>((orow < 256))))
  {
    for (maxIdx = 0; maxIdx < 2; maxIdx++) {
      for (i0 = 0; i0 < 2; i0++) {
        newIm[i0 + (maxIdx << 1)] = inpImg_padded_shared[(static_cast<int32_T>
          (threadIdx.x) + i0) + 33 * (static_cast<int32_T>(threadIdx.y) + maxIdx)];
      }
    }

    for (iter = 0; iter < 2; iter++) {
      startIdx = 1 + iter;
      minIdx = startIdx;
      t = newIm[startIdx - 1];
      i0 = 4 - (iter + startIdx);
      for (b_iter = 0; b_iter < i0; b_iter++) {
        c_iter = static_cast<int32_T>(((static_cast<uint32_T>(startIdx) +
          static_cast<uint32_T>(b_iter)) + 1U));
        if (static_cast<int32_T>(newIm[c_iter - 1]) < static_cast<int32_T>(t)) {
          t = newIm[c_iter - 1];
          minIdx = c_iter;
        }
      }

      t = newIm[minIdx - 1];
      newIm[minIdx - 1] = newIm[iter];
      newIm[iter] = t;
      maxIdx = startIdx;
      t = newIm[startIdx - 1];
      i0 = 4 - (iter + startIdx);
      for (b_iter = 0; b_iter < i0; b_iter++) {
        c_iter = static_cast<int32_T>(((static_cast<uint32_T>(startIdx) +
          static_cast<uint32_T>(b_iter)) + 1U));
        if (static_cast<int32_T>(newIm[c_iter - 1]) > static_cast<int32_T>(t)) {
          t = newIm[c_iter - 1];
          maxIdx = c_iter;
        }
      }

      t = newIm[maxIdx - 1];
      newIm[maxIdx - 1] = newIm[3 - iter];
      newIm[3 - iter] = t;
    }

    val = 0.5 * static_cast<real_T>(newIm[1]) + 0.5 * static_cast<real_T>(newIm
      [2]);
    if (val > 0.0) {
      outImg[orow + (ocol << 8)] = static_cast<uint16_T>((val + 0.5));
    } else {
      outImg[orow + (ocol << 8)] = static_cast<uint16_T>(0U);
    }
  }
}

static __global__ __launch_bounds__(512, 1) void noise_removal_gpu_kernel9(const
  uint16_T noisyRGB[393216], uint16_T inpImg_padded[131072])
{
  uint32_T threadId;
  int32_T minIdx;
  int32_T maxIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  minIdx = static_cast<int32_T>((threadId % 256U));
  maxIdx = static_cast<int32_T>(((threadId - static_cast<uint32_T>(minIdx)) /
    256U));
  if (maxIdx < 512) {
    inpImg_padded[minIdx + (maxIdx << 8)] = noisyRGB[262144 + (minIdx + (maxIdx <<
      8))];
  }
}

void noise_removal_gpu(noise_removal_gpuStackData *SD, const uint16_T noisyRGB
  [393216], uint16_T rgbFixed[393216])
{
  int32_T iy;
  int32_T j;
  int32_T initAuxVar;
  uint16_T (*gpu_noisyRGB)[393216];
  uint16_T (*gpu_inpImg_padded)[131072];
  uint16_T (*b_gpu_inpImg_padded)[131841];
  uint16_T (*gpu_outImg)[131072];
  uint16_T (*b_gpu_outImg)[131072];
  uint16_T (*gpu_rgbFixed)[393216];
  boolean_T outImg_dirtyOnGpu;
  boolean_T b_outImg_dirtyOnGpu;
  boolean_T rgbFixed_dirtyOnCpu;
  hipMalloc(&b_gpu_outImg, 262144UL);
  hipMalloc(&gpu_rgbFixed, 786432UL);
  hipMalloc(&gpu_outImg, 262144UL);
  hipMalloc(&b_gpu_inpImg_padded, 263682UL);
  hipMalloc(&gpu_inpImg_padded, 262144UL);
  hipMalloc(&gpu_noisyRGB, 786432UL);
  rgbFixed_dirtyOnCpu = false;

  /*  Extract the individual red, green, and blue color channels. */
  /*  Median Filter the channels */
  hipMemcpy(gpu_noisyRGB, (void *)&noisyRGB[0], 786432UL,
             hipMemcpyHostToDevice);
  noise_removal_gpu_kernel1<<<dim3(256U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_noisyRGB, *gpu_inpImg_padded);
  noise_removal_gpu_kernel2<<<dim3(1U, 1U, 1U), dim3(288U, 1U, 1U)>>>
    (*b_gpu_inpImg_padded);
  noise_removal_gpu_kernel3<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_inpImg_padded, *b_gpu_inpImg_padded);
  noise_removal_gpu_kernel4<<<dim3(8U, 16U, 1U), dim3(32U, 32U, 1U)>>>
    (*b_gpu_inpImg_padded, *gpu_outImg);
  outImg_dirtyOnGpu = true;
  noise_removal_gpu_kernel5<<<dim3(256U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_noisyRGB, *gpu_inpImg_padded);
  noise_removal_gpu_kernel6<<<dim3(1U, 1U, 1U), dim3(288U, 1U, 1U)>>>
    (*b_gpu_inpImg_padded);
  noise_removal_gpu_kernel7<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_inpImg_padded, *b_gpu_inpImg_padded);
  noise_removal_gpu_kernel8<<<dim3(8U, 16U, 1U), dim3(32U, 32U, 1U)>>>
    (*b_gpu_inpImg_padded, *b_gpu_outImg);
  b_outImg_dirtyOnGpu = true;
  noise_removal_gpu_kernel9<<<dim3(256U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_noisyRGB, *gpu_inpImg_padded);
  noise_removal_gpu_kernel10<<<dim3(1U, 1U, 1U), dim3(288U, 1U, 1U)>>>
    (*b_gpu_inpImg_padded);
  noise_removal_gpu_kernel11<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_inpImg_padded, *b_gpu_inpImg_padded);
  noise_removal_gpu_kernel12<<<dim3(8U, 16U, 1U), dim3(32U, 32U, 1U)>>>
    (*b_gpu_inpImg_padded, *gpu_inpImg_padded);

  /*  Reconstruct the noise free RGB image */
  iy = -1;
  for (j = 0; j < 131072; j++) {
    iy = j;
    if (outImg_dirtyOnGpu) {
      hipMemcpy(&SD->f0.outImg[0], gpu_outImg, 262144UL, hipMemcpyDeviceToHost);
      outImg_dirtyOnGpu = false;
    }

    rgbFixed[j] = SD->f0.outImg[j];
    rgbFixed_dirtyOnCpu = true;
  }

  initAuxVar = iy;
  for (j = 0; j < 131072; j++) {
    iy = (initAuxVar + j) + 1;
    if (b_outImg_dirtyOnGpu) {
      hipMemcpy(&SD->f0.b_outImg[0], b_gpu_outImg, 262144UL,
                 hipMemcpyDeviceToHost);
      b_outImg_dirtyOnGpu = false;
    }

    rgbFixed[iy] = SD->f0.b_outImg[j];
    rgbFixed_dirtyOnCpu = true;
  }

  if (rgbFixed_dirtyOnCpu) {
    hipMemcpy(gpu_rgbFixed, &rgbFixed[0], 786432UL, hipMemcpyHostToDevice);
  }

  noise_removal_gpu_kernel13<<<dim3(256U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_inpImg_padded, iy, *gpu_rgbFixed);
  hipMemcpy(&rgbFixed[0], gpu_rgbFixed, 786432UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_noisyRGB);
  hipFree(*gpu_inpImg_padded);
  hipFree(*b_gpu_inpImg_padded);
  hipFree(*gpu_outImg);
  hipFree(*gpu_rgbFixed);
  hipFree(*b_gpu_outImg);
}

/* End of code generation (noise_removal_gpu.cu) */
